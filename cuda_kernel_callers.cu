#include "hip/hip_runtime.h"
#include <hipblas.h>
#pragma comment (lib, "cublas.lib")

#include "cuda_kernel_callers.cuh"

void cuda_simple_dgemm(
   const real* a, const real* b, real* c,
   size_t M, size_t N, size_t K)
{
	const size_t thread_per_block_1_dim = 32;
	dim3 thread_per_block(thread_per_block_1_dim, thread_per_block_1_dim);	
	dim3 blocks(
		std::ceil(static_cast<float>(N) / thread_per_block_1_dim),
		std::ceil(static_cast<float>(M) / thread_per_block_1_dim));

	cuda_simple_dgemm_kernel<<<blocks, thread_per_block>>>(a, b, c, M, N, K);
}

void cublas_dgemm(
   const real* a, const real* b, real* c,
   size_t M, size_t N, size_t K)
{
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const real alpha = 1;
	const real beta = 0;

	hipblasSgemm(
		handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
		M, N, K,
		&alpha,
		a, M,
		b, K,
		&beta,
		c, M);

	hipblasDestroy(handle);
} 